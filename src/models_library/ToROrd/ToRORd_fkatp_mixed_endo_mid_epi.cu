#include "hip/hip_runtime.h"
#include "ToRORd_fkatp_mixed_endo_mid_epi.h"
#include <stddef.h>
#include <stdint.h>

__constant__  size_t pitch;
__constant__  real abstol;
__constant__  real reltol;
__constant__  real max_dt;
__constant__  real min_dt;
__constant__  uint8_t use_adpt;

size_t pitch_h;

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    uint8_t use_adpt_h = (uint8_t)solver->adaptive;

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(use_adpt), &use_adpt_h, sizeof(uint8_t)));
    log_info("Using ToRORd_fkatp_mixed_endo_mid_epi GPU model\n");

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_h) {
        real reltol_h = solver->rel_tol;
        real abstol_h = solver->abs_tol;
        real max_dt_h = solver->max_dt;
        real min_dt_h = solver->min_dt;

        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(reltol), &reltol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(abstol), &abstol_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(max_dt), &max_dt_h, sizeof(real)));
        check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(min_dt), &min_dt_h, sizeof(real)));
        log_info("Using Adaptive Euler model to solve the ODEs\n");
    } else {
        log_info("Using Euler model to solve the ODEs\n");
    }

    // Execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_h)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    // Get initial condition from extra_data
    real *initial_conditions_endo = NULL;
    real *initial_conditions_epi = NULL;
    real *initial_conditions_mid = NULL;
    real *mapping = NULL;
    real *initial_conditions_endo_device = NULL;
    real *initial_conditions_epi_device = NULL;
    real *initial_conditions_mid_device = NULL;
    real *mapping_device = NULL;

    if(solver->ode_extra_data) 
    {
        initial_conditions_endo = (real *)solver->ode_extra_data;
        initial_conditions_epi = (real *)solver->ode_extra_data+NEQ;
        initial_conditions_mid = (real *)solver->ode_extra_data+NEQ+NEQ;
        mapping = (real *)solver->ode_extra_data+NEQ+NEQ+NEQ;
        check_cuda_error(hipMalloc((void **)&initial_conditions_endo_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_endo_device, initial_conditions_endo, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_epi_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_epi_device, initial_conditions_epi, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_mid_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_mid_device, initial_conditions_mid, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&mapping_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(mapping_device, mapping, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
    }
    else
    {
        log_error_and_exit("You must supply a mask function to tag the cells when using this mixed model!\n");
    }

    kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv,\
                                                            initial_conditions_endo_device, initial_conditions_epi_device, initial_conditions_mid_device,\
                                                            mapping_device, num_volumes);

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();

    check_cuda_error(hipFree(initial_conditions_endo_device));
    check_cuda_error(hipFree(initial_conditions_epi_device));
    check_cuda_error(hipFree(initial_conditions_mid_device));
    check_cuda_error(hipFree(mapping_device));

    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adaptive mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    // Get the mapping array
    uint32_t num_volumes = ode_solver->original_num_cells;
    real *mapping = NULL;
    real *mapping_device = NULL;
    if(ode_solver->ode_extra_data) 
    {
        mapping = (real *)ode_solver->ode_extra_data+NEQ+NEQ+NEQ;
        check_cuda_error(hipMalloc((void **)&mapping_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(mapping_device, mapping, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
    }
    else 
    {
        log_error_and_exit("You must supply a mask function to tag the cells when using this mixed model!\n");
    }

    solve_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps, mapping_device);

    check_cuda_error(hipPeekAtLastError());

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if (mapping_device) check_cuda_error(hipFree(mapping_device));

}

__global__ void kernel_set_model_initial_conditions(real *sv,\
                                                real *initial_endo, real *initial_epi, real *initial_mid,\
                                                real *mapping, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) 
    {
        for (int i = 0; i < NEQ; i++)
        {
            if (mapping[threadID] == 0.0)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_endo[i];
            else if (mapping[threadID] == 1.0)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_epi[i];
            else
                *((real * )((char *) sv + pitch * i) + threadID) = initial_mid[i];
        }
            
        if(use_adpt) 
        {
            *((real *)((char *)sv + pitch * 43) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 44) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 45) + threadID) = 0.0;    // previous dt
        }
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real cur_time, real dt, real *sv, real* stim_currents,
                            uint32_t *cells_to_solve, uint32_t num_cells_to_solve, int num_steps, real *mapping)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                RHS_gpu(sv, rDY, stim_currents[threadID], mapping[threadID], sv_id, dt);

                for(int i = 0; i < NEQ; i++) {
                    *((real *)((char *)sv + pitch * i) + sv_id) =
                        dt * rDY[i] + *((real *)((char *)sv + pitch * i) + sv_id);
                }
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], mapping[threadID], cur_time + max_dt, sv_id);
        }
    }
}

inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real mapping, real final_time, int thread_id)
{

    #define DT *((real *)((char *)sv + pitch * 43) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * 44) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * 45) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = pow(abstol, 2.0f);

    // dt = ((time_new + dt) > final_time) ? (final_time - time_new) : dt;
    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, mapping, thread_id, dt);
    time_new += dt;

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

    int count = 0;

    int count_limit = (final_time - time_new) / min_dt;

    int aux_count_limit = count_limit + 2000000;

    if(aux_count_limit > 0) {
        count_limit = aux_count_limit;
    }

    while(1) {

        for(int i = 0; i < NEQ; i++) {
            // stores the old variables in a vector
            edos_old_aux_[i] = sv_local[i];
            // //computes euler method
            edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
            // steps ahead to compute the rk2 method
            sv_local[i] = edos_new_euler_[i];
        }

        time_new += dt;

        RHS_gpu(sv_local, rDY, stim_curr, mapping, thread_id, dt);
        time_new -= dt; // step back

        real greatestError = 0.0, auxError = 0.0;
        //#pragma unroll
        for(int i = 0; i < NEQ; i++) {

            // stores the new evaluation
            _k2__[i] = rDY[i];
            _aux_tol = fabs(edos_new_euler_[i]) * reltol;
            _tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

            // finds the greatest error between  the steps
            auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

            greatestError = (auxError > greatestError) ? auxError : greatestError;
        }

        /// adapt the time step
        greatestError += __tiny_;
        previous_dt = dt;
        /// adapt the time step
        dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

        if(time_new + dt > final_time) {
            dt = final_time - time_new;
        }

        // it doesn't accept the solution
        if(count < count_limit && (greatestError >= 1.0f)) {
            // restore the old values to do it again
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_old_aux_[i];
            }
            count++;
            // throw the results away and compute again
        } else {
            count = 0;

            // if(greatestError >=1.0) {
            //    printf("Thread //d,accepting solution with error > //lf \n", threadID, greatestError);
            //}

            // it accepts the solutions
            // int aux = (dt > max_step && max_step != 0);
            // dt = (aux) ? max_step : dt;

            if(dt < min_dt) {
                dt = min_dt;
            }

            else if(dt > max_dt && max_dt != 0) {
                dt = max_dt;
            }

            if(time_new + dt > final_time) {
                dt = final_time - time_new;
            }

            // change vectors k1 e k2 , para que k2 seja aproveitado como k1 na proxima iteração
            //#pragma unroll
            for(int i = 0; i < NEQ; i++) {
                _k_aux__[i] = _k2__[i];
                _k2__[i] = _k1__[i];
                _k1__[i] = _k_aux__[i];
            }

            // it steps the method ahead, with euler solution
            //#pragma unroll
            for(int i = 0; i < NEQ; i++) {
                sv_local[i] = edos_new_euler_[i];
            }

            // verifica se o incremento para a próxima iteração ultrapassa o tempo de salvar, q neste caso é o tempo
            // final
            if(time_new + previous_dt >= final_time) {
                // se são iguais, ja foi calculada a iteração no ultimo passo de tempo e deve-se para o laço
                // nao usar igualdade - usar esta conta, pode-se mudar a tolerância
                // printf("//d: //lf\n", threadID, fabs(final_time - time_new));
                if((fabs(final_time - time_new) < 1.0e-5)) {
                    break;
                } else if(time_new < final_time) {
                    dt = previous_dt = final_time - time_new;
                    time_new += previous_dt;
                    break;
                } else {
                    dt = previous_dt = min_dt;
                    time_new += (final_time - time_new);
                    printf("Nao era pra chegar aqui: %d: %lf\n", thread_id, final_time - time_new);
                    break;
                }
            } else {
                time_new += previous_dt;
            }
        }
    }

    //#pragma unroll
    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, real mapping, int threadID_, real dt) 
{
    // Get the celltype for the current cell
    real celltype = mapping;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // State variables
    real STATES[NEQ];
    if (use_adpt)
    {
        for (uint32_t i = 0; i < NEQ; i++)
            STATES[i] = sv[i];
    }
    else
    {
        for (uint32_t i = 0; i < NEQ; i++)
            STATES[i] = *((real *)((char *)sv + pitch * i) + threadID_);
    }

    #include "ToRORd_fkatp_mixed_endo_mid_epi.common.c"
}
