#include "hip/hip_runtime.h"
#include "trovato_2019.h"
#include <stddef.h>
#include <stdint.h>

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {

        real STATES[NEQ];
        
        // Steady-state 40 pulses (BCL=1000ms)
        STATES[0] = -86.7099;
        STATES[1] = 0.005431;
        STATES[2] = 0.000104;
        STATES[3] = 8.25533;
        STATES[4] = 8.25502;
        STATES[5] = 8.25503;
        STATES[6] = 143.743;
        STATES[7] = 143.744;
        STATES[8] = 143.744;
        STATES[9] = 4.4e-05;
        STATES[10] = 0.000103;
        STATES[11] = 1.26947;
        STATES[12] = 1.25254;
        STATES[13] = 1.27103;
        STATES[14] = 1.1e-05;
        STATES[15] = 0;
        STATES[16] = 0.006303;
        STATES[17] = 0.789469;
        STATES[18] = 0.789392;
        STATES[19] = 0.791301;
        STATES[20] = 0.580955;
        STATES[21] = 0.791719;
        STATES[22] = 0.000241;
        STATES[23] = 0.463851;
        STATES[24] = 0.239936;
        STATES[25] = 0.000272;
        STATES[26] = 0.646362;
        STATES[27] = 0.98999;
        STATES[28] = 0;
        STATES[29] = 1;
        STATES[30] = 0.926919;
        STATES[31] = 1;
        STATES[32] = 1;
        STATES[33] = 0.999976;
        STATES[34] = 1;
        STATES[35] = 1;
        STATES[36] = 0.005885;
        STATES[37] = 0.000303;
        STATES[38] = 0.994251;
        STATES[39] = 0.000367;
        STATES[40] = 0.566131;
        STATES[41] = 0.189842;
        STATES[42] = 0.000222;
        STATES[43] = 0.233515;
        STATES[44] = 0.997077;
        STATES[45] = 0.471259;

        for (int i = 0; i < NEQ; i++)
            *((real * )((char *) sv + pitch * i) + threadID) = STATES[i];


        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * NEQ) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * (NEQ + 1)) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * (NEQ + 2)) + threadID) = 0.0;    // previous dt
        }
    }
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {

    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // State variables
    real V;
    real CaMKt;
    real cass;
    real nai;
    real nasl;
    real nass;
    real ki;
    real kss;
    real ksl;
    real cai;
    real casl;
    real cansr;
    real cajsr;
    real cacsr;
    real Jrel1;
    real Jrel2;
    real m;
    real hf;
    real hs;
    real j;
    real hsp;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real i1;
    real i2;
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real ffp;
    real fcafp;
    real nca;
    real b;
    real g;
    real xrf;
    real xrs;
    real xs1;
    real xs2;
    real y;
    real xk1;
    real u;   

    if (use_adpt_dt)
    {
        V = sv[0];
        CaMKt = sv[1];
        cass = sv[2];
        nai = sv[3];
        nasl = sv[4];
        nass = sv[5];
        ki = sv[6];
        kss = sv[7];
        ksl = sv[8];
        cai = sv[9];
        casl = sv[10];
        cansr = sv[11];
        cajsr = sv[12];
        cacsr = sv[13];
        Jrel1 = sv[14];
        Jrel2 = sv[15];
        m = sv[16];
        hf = sv[17];
        hs = sv[18];
        j = sv[19];
        hsp = sv[20];
        jp = sv[21];
        mL = sv[22];
        hL = sv[23];
        hLp = sv[24];
        a = sv[25];
        i1 = sv[26];
        i2 = sv[27];
        d = sv[28];
        ff = sv[29];
        fs = sv[30];
        fcaf = sv[31];
        fcas = sv[32];
        jca = sv[33];
        ffp = sv[34];
        fcafp = sv[35];
        nca = sv[36];
        b = sv[37];
        g = sv[38];
        xrf = sv[39];
        xrs = sv[40];
        xs1 = sv[41];
        xs2 = sv[42];
        y = sv[43];
        xk1 = sv[44];
        u = sv[45];
    }
    else
    {
        V = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt = *((real *)((char *)sv + pitch * 1) + threadID_);
        cass = *((real *)((char *)sv + pitch * 2) + threadID_);
        nai = *((real *)((char *)sv + pitch * 3) + threadID_);
        nasl = *((real *)((char *)sv + pitch * 4) + threadID_);
        nass = *((real *)((char *)sv + pitch * 5) + threadID_);
        ki = *((real *)((char *)sv + pitch * 6) + threadID_);
        kss = *((real *)((char *)sv + pitch * 7) + threadID_);
        ksl = *((real *)((char *)sv + pitch * 8) + threadID_);
        cai = *((real *)((char *)sv + pitch * 9) + threadID_);
        casl = *((real *)((char *)sv + pitch * 10) + threadID_);
        cansr = *((real *)((char *)sv + pitch * 11) + threadID_);
        cajsr = *((real *)((char *)sv + pitch * 12) + threadID_);
        cacsr = *((real *)((char *)sv + pitch * 13) + threadID_);
        Jrel1 = *((real *)((char *)sv + pitch * 14) + threadID_);
        Jrel2 = *((real *)((char *)sv + pitch * 15) + threadID_);
        m = *((real *)((char *)sv + pitch * 16) + threadID_);
        hf = *((real *)((char *)sv + pitch * 17) + threadID_);
        hs = *((real *)((char *)sv + pitch * 18) + threadID_);
        j = *((real *)((char *)sv + pitch * 19) + threadID_);
        hsp = *((real *)((char *)sv + pitch * 20) + threadID_);
        jp = *((real *)((char *)sv + pitch * 21) + threadID_);
        mL = *((real *)((char *)sv + pitch * 22) + threadID_);
        hL = *((real *)((char *)sv + pitch * 23) + threadID_);
        hLp = *((real *)((char *)sv + pitch * 24) + threadID_);
        a = *((real *)((char *)sv + pitch * 25) + threadID_);
        i1 = *((real *)((char *)sv + pitch * 26) + threadID_);
        i2 = *((real *)((char *)sv + pitch * 27) + threadID_);
        d = *((real *)((char *)sv + pitch * 28) + threadID_);
        ff = *((real *)((char *)sv + pitch * 29) + threadID_);
        fs = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcaf = *((real *)((char *)sv + pitch * 31) + threadID_);
        fcas = *((real *)((char *)sv + pitch * 32) + threadID_);
        jca = *((real *)((char *)sv + pitch * 33) + threadID_);
        ffp = *((real *)((char *)sv + pitch * 34) + threadID_);
        fcafp = *((real *)((char *)sv + pitch * 35) + threadID_);
        nca = *((real *)((char *)sv + pitch * 36) + threadID_);
        b = *((real *)((char *)sv + pitch * 37) + threadID_);
        g = *((real *)((char *)sv + pitch * 38) + threadID_);
        xrf = *((real *)((char *)sv + pitch * 39) + threadID_);
        xrs = *((real *)((char *)sv + pitch * 40) + threadID_);
        xs1 = *((real *)((char *)sv + pitch * 41) + threadID_);
        xs2 = *((real *)((char *)sv + pitch * 42) + threadID_);
        y = *((real *)((char *)sv + pitch * 43) + threadID_);
        xk1 = *((real *)((char *)sv + pitch * 44) + threadID_);
        u = *((real *)((char *)sv + pitch * 45) + threadID_);
    }

    #include "trovato_2019_common.inc"
}

//Include the default solver used by all models.
#include "../default_solvers.cu"
